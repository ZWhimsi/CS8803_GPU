
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <limits.h>
#include <math.h>
#include <float.h>
#include <string.h>

typedef int DTYPE;

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n", \
                    __FILE__, __LINE__, error, hipGetErrorString(error), #call); \
            exit(1); \
        } \
    } while (0)

// Logging macros for better debugging
#define LOG_INFO(fmt, ...) printf("[INFO] " fmt "\n", ##__VA_ARGS__)
#define LOG_DEBUG(fmt, ...) printf("[DEBUG] " fmt "\n", ##__VA_ARGS__)
#define LOG_ERROR(fmt, ...) fprintf(stderr, "[ERROR] " fmt "\n", ##__VA_ARGS__)
#define LOG_STAGE(stage) printf("\n[STAGE] %s\n", stage)

// GPU Kernel for bitonic sort with global memory
__global__ void BitonicSort_global(int* data, int j, int k, int size){
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if (i >= size) return;
  
  int partnerGlobalIdx = i ^ j;
  
  if (i < partnerGlobalIdx && i < size && partnerGlobalIdx < size) {
    bool ascending = (i & k) == 0; 
    
    // Coalesced memory access pattern
    int val1 = data[i];
    int val2 = data[partnerGlobalIdx];

    // Compare and swap if needed
    if ((val1 > val2) == ascending) {
      data[i] = val2;
      data[partnerGlobalIdx] = val1;
    }
  }
}

// GPU Kernel for bitonic sort with shared memory
__global__ void BitonicSort_shared(int* data, int j, int k, int size){
  extern __shared__ DTYPE s_array[];
  
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  
  // Handle threads beyond array size
  if (i >= size) return;
  
  // Load data into shared memory
  s_array[threadIdx.x] = data[i];
  __syncthreads();
  
  int partnerGlobalIdx = i ^ j;
  int partnerBlockIdx = partnerGlobalIdx / blockDim.x;
  int partnerLocalIdx = partnerGlobalIdx % blockDim.x;
  bool sameBlock = (partnerBlockIdx == blockIdx.x);
  
  if (i < partnerGlobalIdx && i < size && partnerGlobalIdx < size) {
    bool ascending = (i & k) == 0; 
    
    int val1, val2;
    if (sameBlock) {
      // Both elements are in the same block
      val1 = s_array[threadIdx.x];
      val2 = s_array[partnerLocalIdx];
    } else {
      // Partner is in a different block, access global memory
      val1 = s_array[threadIdx.x];
      val2 = data[partnerGlobalIdx];
    }
    
    // Compare and swap if needed
    if ((val1 > val2) == ascending) {
      if (sameBlock) {
        s_array[threadIdx.x] = val2;
        s_array[partnerLocalIdx] = val1;
      } else {
        s_array[threadIdx.x] = val2;
        data[partnerGlobalIdx] = val1;
      }
    }
  }
  
  __syncthreads();
  
  // Write back to global memory
  if (i < size) {
    data[i] = s_array[threadIdx.x];
  }
}

// Helper function to print array
void printArray(int* arr, int size, const char* prefix = "") {
    printf("%s[", prefix);
    int print_limit = (size > 20) ? 20 : size;
    for (int i = 0; i < print_limit; i++) {
        printf("%d", arr[i]);
        if (i < print_limit - 1) printf(", ");
    }
    if (size > 20) {
        printf("... (%d more elements)", size - 20);
    }
    printf("]\n");
}

// Check if a number is a power of 2
bool isPowerOfTwo(int n) {
    return n && !(n & (n - 1));
}

// Find the next power of 2 greater than or equal to n
int nextPowerOfTwo(int n) {
    if (isPowerOfTwo(n)) return n;
    int power = 1;
    while (power < n) {
        power *= 2;
    }
    return power;
}

// CPU implementation of bitonic sort for validation
void bitonicSortCPU(int* arr, int size) {
    for (int k = 2; k <= size; k *= 2) {
        for (int j = k/2; j > 0; j /= 2) {
            for (int i = 0; i < size; i++) {
                int partnerIdx = i ^ j;
                if (i < partnerIdx && partnerIdx < size) {
                    bool ascending = (i & k) == 0;
                    if ((arr[i] > arr[partnerIdx]) == ascending) {
                        int temp = arr[i];
                        arr[i] = arr[partnerIdx];
                        arr[partnerIdx] = temp;
                    }
                }
            }
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        LOG_ERROR("Usage: %s <array_size>", argv[0]);
        return 1;
    }
    
    int original_size = atoi(argv[1]);
    if (original_size <= 0) {
        LOG_ERROR("Array size must be positive");
        return 1;
    }
    
    LOG_INFO("Starting bitonic sort with array size: %d", original_size);
    
    // For bitonic sort, size must be a power of 2
    int size = nextPowerOfTwo(original_size);
    bool needs_padding = (size != original_size);
    
    if (needs_padding) {
        LOG_INFO("Padding array from %d to %d elements (next power of 2)", original_size, size);
    }
    
    // Allocate host memory for the array
    int* arrCpu = (int*)malloc(size * sizeof(int));
    if (!arrCpu) {
        LOG_ERROR("Failed to allocate memory for CPU array");
        return 1;
    }
    
    // Initialize array with random values
    LOG_STAGE("Generating random array");
    srand(time(NULL));
    for (int i = 0; i < original_size; i++) {
        arrCpu[i] = rand() % 1000;
    }
    
    // Pad with INT_MAX if needed
    for (int i = original_size; i < size; i++) {
        arrCpu[i] = INT_MAX;
    }
    
    LOG_INFO("Original array:");
    printArray(arrCpu, original_size, "  ");
    
    // Allocate memory for GPU result using pinned memory for faster D2H
    int* arrSortedGpu = nullptr;
    hipError_t pinned_result = hipHostMalloc(&arrSortedGpu, size * sizeof(int), hipHostMallocDefault);
    bool use_pinned_output = (pinned_result == hipSuccess);
    
    if (!use_pinned_output) {
        LOG_DEBUG("Pinned memory for output failed, using regular memory");
        arrSortedGpu = (int*)malloc(size * sizeof(int));
        if (!arrSortedGpu) {
            LOG_ERROR("Failed to allocate memory for GPU result array");
            free(arrCpu);
            return 1;
        }
    } else {
        LOG_DEBUG("Using pinned memory for output (faster D2H transfers)");
    }

    LOG_STAGE("Setting up GPU memory and data transfer");

    // Transfer data (arr_cpu) to device
    DTYPE* d_arr;
    
    // Try pinned memory for input
    int* h_arr_pinned = nullptr;
    hipError_t input_pinned = hipHostMalloc(&h_arr_pinned, size * sizeof(int), hipHostMallocDefault);
    bool use_pinned_input = (input_pinned == hipSuccess);
    
    if (use_pinned_input) {
        memcpy(h_arr_pinned, arrCpu, size * sizeof(int));
        LOG_DEBUG("Using pinned memory for input");
    } else {
        LOG_DEBUG("Using regular memory for input");
    }
    
    CUDA_CHECK(hipMalloc(&d_arr, size * sizeof(DTYPE)));

    // Copy data from host to device
    LOG_DEBUG("Copying data from host to device");
    CUDA_CHECK(hipMemcpy(d_arr, use_pinned_input ? h_arr_pinned : arrCpu, 
                          size * sizeof(DTYPE), hipMemcpyHostToDevice));
    LOG_INFO("Data successfully copied to GPU");

    /* ==== DO NOT MODIFY CODE BELOW THIS LINE ==== */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float h2dTime, kernelTime, d2hTime;

    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(d_arr, use_pinned_input ? h_arr_pinned : arrCpu, 
                          size * sizeof(DTYPE), hipMemcpyHostToDevice));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&h2dTime, start, stop);

    hipEventRecord(start);
    /* ==== DO NOT MODIFY CODE ABOVE THIS LINE ==== */

    // Perform bitonic sort on GPU using shared memory
    LOG_STAGE("Starting shared memory bitonic sort on GPU");
    // Optimal launch configuration for H100
    int threadsPerBlock = 512;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    LOG_INFO("Launch configuration: %d blocks, %d threads per block", blocksPerGrid, threadsPerBlock);

    // Calculate shared memory size needed
    size_t sharedMemSize = threadsPerBlock * sizeof(DTYPE);
    LOG_DEBUG("Shared memory size: %zu bytes", sharedMemSize);

    // Use smart hybrid approach: shared memory for small arrays, global memory for large arrays
    int step_count = 0;
    if (size <= 512) {
        // Small arrays: Use shared memory kernel (benefits from shared memory)
        LOG_DEBUG("Using shared memory kernel for small array");
        for (int k = 2; k <= size; k *= 2) {
            for (int j = k/2; j > 0; j /= 2) {
                BitonicSort_shared<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_arr, j, k, size);
                step_count++;
                LOG_DEBUG("Step %d: Using shared memory kernel (j=%d, k=%d)", step_count, j, k);
            }
        }
        LOG_INFO("Shared memory bitonic sort completed in %d steps", step_count);
    } else {
        // Large arrays: Use global memory kernel
        LOG_DEBUG("Using global memory kernel for large array");
        for (int k = 2; k <= size; k *= 2) {
            for (int j = k/2; j > 0; j /= 2) {
                BitonicSort_global<<<blocksPerGrid, threadsPerBlock>>>(d_arr, j, k, size);
                step_count++;
            }
        }
        LOG_INFO("Global memory bitonic sort completed in %d steps", step_count);
    }

    CUDA_CHECK(hipDeviceSynchronize());
    LOG_DEBUG("Shared memory kernel completed");
    LOG_INFO("Shared memory bitonic sort completed");

    /* ==== DO NOT MODIFY CODE BELOW THIS LINE ==== */
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kernelTime, start, stop);

    hipEventRecord(start);
    /* ==== DO NOT MODIFY CODE ABOVE THIS LINE ==== */

    // Transfer sorted data back to host (copied to arrSortedGpu)
    LOG_STAGE("Transferring sorted data back to host");
    
    // OPTIMIZATION: Use async copy if we have pinned memory
    if (use_pinned_output) {
        hipStream_t stream;
        CUDA_CHECK(hipStreamCreate(&stream));
        CUDA_CHECK(hipMemcpyAsync(arrSortedGpu, d_arr, size * sizeof(DTYPE), 
                                   hipMemcpyDeviceToHost, stream));
        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipStreamDestroy(stream));
    } else {
        CUDA_CHECK(hipMemcpy(arrSortedGpu, d_arr, size * sizeof(DTYPE), hipMemcpyDeviceToHost));
    }
    
    LOG_INFO("Sorted array:");
    printArray(arrSortedGpu, original_size, "  ");

    /* ==== DO NOT MODIFY CODE BELOW THIS LINE ==== */
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&d2hTime, start, stop);
    /* ==== DO NOT MODIFY CODE ABOVE THIS LINE ==== */

    // CPU sort for validation
    LOG_STAGE("Performing CPU sort for comparison");
    int* arrCpuSorted = (int*)malloc(size * sizeof(int));
    memcpy(arrCpuSorted, arrCpu, size * sizeof(int));
    
    clock_t cpuStart = clock();
    bitonicSortCPU(arrCpuSorted, size);
    clock_t cpuEnd = clock();
    double cpuTime = ((double)(cpuEnd - cpuStart)) / CLOCKS_PER_SEC * 1000.0;

    // Validate results (only compare original_size elements)
    LOG_STAGE("Validating results");
    bool valid = true;
    for (int i = 0; i < original_size; i++) {
        if (arrSortedGpu[i] != arrCpuSorted[i]) {
            LOG_ERROR("Mismatch at index %d: GPU=%d, CPU=%d", i, arrSortedGpu[i], arrCpuSorted[i]);
            valid = false;
            
            // Show context around mismatch
            int start = (i > 5) ? i - 5 : 0;
            int end = (i + 5 < original_size) ? i + 5 : original_size;
            
            printf("[DEBUG] GPU array around mismatch: [");
            for (int j = start; j < end; j++) {
                printf("%d", arrSortedGpu[j]);
                if (j < end - 1) printf(", ");
            }
            printf("]\n");
            
            printf("[DEBUG] CPU array around mismatch: [");
            for (int j = start; j < end; j++) {
                printf("%d", arrCpuSorted[j]);
                if (j < end - 1) printf(", ");
            }
            printf("]\n");
            
            LOG_ERROR("Validation failed - results don't match at index %d", i);
            break;
        }
    }
    
    if (valid) {
        LOG_INFO("Validation successful - GPU and CPU results match!");
        LOG_INFO("FUNCTIONAL SUCCESS");
        printf("FUNCTIONAL SUCCESS\n");
        
        double gpuTime = h2dTime + kernelTime + d2hTime;
        double elementsPerSecond = (original_size / (gpuTime / 1000.0)) / 1e6;
        
        printf("Array size         : %d\n", original_size);
        printf("CPU Sort Time (ms) : %f\n", cpuTime);
        printf("GPU Sort Time (ms) : %f\n", gpuTime);
        printf("GPU Sort Speed     : %f million elements per second\n", elementsPerSecond);
        
        if (elementsPerSecond > 1000) {
            printf("PERF PASSING\n");
        } else {
            printf("PERF FAILING (need > 1000 MOPE/s, got %.2f)\n", elementsPerSecond);
        }
        
        printf("GPU Sort is %3.0fx faster than CPU !!!\n", cpuTime / gpuTime);
        printf("H2D Transfer Time (ms): %f\n", h2dTime);
        printf("Kernel Time (ms)      : %f\n", kernelTime);
        printf("D2H Transfer Time (ms): %f\n", d2hTime);
        
        // Debug info
        printf("\nOptimization Status:\n");
        printf("- Input pinned memory: %s\n", use_pinned_input ? "YES" : "NO");
        printf("- Output pinned memory: %s\n", use_pinned_output ? "YES" : "NO");
    } else {
        LOG_ERROR("FUNCTIONAL FAIL");
    }

    // Clean up GPU memory
    CUDA_CHECK(hipFree(d_arr));
    if (use_pinned_input) {
        CUDA_CHECK(hipHostFree(h_arr_pinned));
    }
    if (use_pinned_output) {
        CUDA_CHECK(hipHostFree(arrSortedGpu));
    } else {
        free(arrSortedGpu);
    }
    
    free(arrCpu);
    free(arrCpuSorted);
    
    return valid ? 0 : 1;
}
