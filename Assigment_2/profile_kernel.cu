#include <hip/hip_runtime_api.h>

// Add this function to get detailed kernel metrics
void profileKernel() {
    // Warmup run
    int size = 134217728;
    int* d_arr;
    hipMalloc(&d_arr, size * sizeof(int));
    
    // Initialize with test data
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
    // Profile specific kernels
    hipProfilerStart();
    
    // Test different j values to see memory access patterns
    int test_j_values[] = {1, 2, 4, 8, 16, 32, 64, 128, 256, 512, 1024, 2048, 4096};
    
    for (int i = 0; i < 13; i++) {
        int j = test_j_values[i];
        int k = j * 2;
        
        // Time the kernel
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipEventRecord(start);
        BitonicSort_global<<<blocksPerGrid, threadsPerBlock>>>(d_arr, j, k, size);
        hipEventRecord(stop);
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        
        printf("j=%d: %.3f ms\n", j, milliseconds);
        
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    
    hipProfilerStop();
    hipFree(d_arr);
}
