#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    printf("Testing basic CUDA functionality...\n\n");
    
    // 1. Check CUDA device
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("hipGetDeviceCount FAILED: %s\n", hipGetErrorString(err));
        return 1;
    }
    printf("Found %d CUDA device(s)\n", deviceCount);
    
    // 2. Set device
    err = hipSetDevice(0);
    if (err != hipSuccess) {
        printf("hipSetDevice FAILED: %s\n", hipGetErrorString(err));
        return 1;
    }
    printf("Successfully set device 0\n");
    
    // 3. Try small allocation
    void* d_ptr;
    size_t size = 1024; // Just 1KB
    printf("\nTrying to allocate 1KB on GPU... ");
    err = hipMalloc(&d_ptr, size);
    if (err != hipSuccess) {
        printf("FAILED\n");
        printf("Error: %s (code %d)\n", hipGetErrorString(err), err);
        
        // Try to reset
        printf("\nAttempting device reset...\n");
        hipDeviceReset();
        
        // Try again after reset
        printf("Trying allocation again after reset... ");
        err = hipMalloc(&d_ptr, size);
        if (err != hipSuccess) {
            printf("STILL FAILED\n");
            printf("Error: %s (code %d)\n", hipGetErrorString(err), err);
            return 1;
        }
    }
    
    printf("SUCCESS\n");
    
    // 4. Try larger allocations
    size_t test_sizes[] = {1024*1024, 10*1024*1024, 100*1024*1024, 512*1024*1024};
    const char* size_names[] = {"1MB", "10MB", "100MB", "512MB"};
    
    for (int i = 0; i < 4; i++) {
        printf("Allocating %s... ", size_names[i]);
        void* test_ptr;
        err = hipMalloc(&test_ptr, test_sizes[i]);
        if (err == hipSuccess) {
            printf("SUCCESS\n");
            hipFree(test_ptr);
        } else {
            printf("FAILED (%s)\n", hipGetErrorString(err));
            break;
        }
    }
    
    // Clean up
    if (d_ptr) hipFree(d_ptr);
    
    printf("\nCUDA test completed.\n");
    return 0;
}
